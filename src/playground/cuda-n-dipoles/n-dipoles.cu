/* 
   n-dipoles simulation on CUDA architecture 
   (c) 2016 Livio and Tiziano Mengotti 

   to compile it: nvcc n-dipoles.cu -o n-dipoles.out
   to execute the example ./n-dipoles.out

   arrangement of arrays is
   index      0 1 2 3 4 5 ...  n-2 n-1
   particle   p e p e p e      p   e      with p proton and e electron
   dipole nb  0 0 1 1 2 2      n-1 n-1    this is the dipole number

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#define Np 1280 // number of particles (dipoles are half of them), should be CUDA core count and even
#define Nd Np/2 // number of dipoles
//TODO: adjust these constants
#define T 0.0001 // timestamp
#define R 5.291772106712E−11  // Bohr radius in meter
#define Q 1.6021773349E-19 //elementar charge in Coulomb
#define Q2 Q*Q
#define PI 3.1415926535
#define COULOMB 1/(4*PI)
#define Mp 1.672623110E-27  // proton mass in kg
#define Me 9.109389754E-31  // electron mass in kg

__device__ double sqr(double x) {
	return x*x;
}

__device__ double getDistanceSquared(int p1, int p2, double *x, double *y) {
	double distance = 0;
	if ((p1<Np) && (p2<Np)) {
		distance = sqr(x[p1]-x[p2])+sqr((y[p1]-y[p2]));		
	}
	return distance;
}


// Projects a magnitude along x and y axis
__device__ void projectVectorXY(double magnitude,
                              double x1, double x2, double y1, double y2, 
                              double *px, double* py) {

	double distance;
	double deltax; double deltay;
	
	deltax = (x2-x1);
	deltay = (y2-y1);
	distance = sqrt(sqr(deltax) + sqr(deltay));	

	// TODO: double check if projection formula is correct
	*px = magnitude * deltax / distance; 
	*py = magnitude * deltay / distance;
	
}  

__device__ void getElectricAcceleration(int p1, int p2, 
                                        double *x, double *y, double *ax, double *ay) {
	
	// acceleration is calculated on p1	
	double acceleration=0;
	double mass;
	if ((p1 % 2) == 0) // even indexes are proton
		mass = Mp;
	else    mass = Me; // odd indexes are electrons
		

	if ((p1<Np) && (p2<Np)) {
		acceleration = COULOMB * Q2 * getDistanceSquared(p1, p2, x, y) / mass;  
		// now we need to project acceleration along (x1-x2) and (y1-y2)
		projectVectorXY(acceleration, x[p1], x[p2], y[p1], y[p2], ax, ay);
	}

}


__global__ void simulate_dipoles(double *x, double *y, double *omega, 
				 double *ax, double *ay, double *angle,
                                 double *E_pot) {
	int tid = blockIdx.x; 
        int iselectron;
	double ax_temp; double ay_temp;

	   if (tid<Np) {
		// 1. calculate acceleration on tid particle
		ax[tid]=0; ay[tid]=0;

		for (int i=0; i<Np; i++) {
			if (i==tid) continue; // we do not calculate on ourself
			iselectron = (i % 2);
			// we do not calculate acceleration on the partner particle on the dipole
			if ((iselectron==1) && (i==tid-1)) continue;
			if ((iselectron==0) && (i==tid+1)) continue;	
			
			getElectricAcceleration(tid, i, x, y, &ax_temp, &ay_temp);
			ax[tid]=ax[tid]+ax_temp;
			ay[tid]=ay[tid]+ay_temp;
		}

		__syncthreads();

		// 2. update omega (angular velocity) with the projected acceleration,
		//    we do it only on half of the cores
		if (tid%2==0) {
			// the axis of projection is perpendicular
			// of (x1,y1)<-->(x2,y2)
		
				
		}
		
		__syncthreads();

		// 3. calculate new x and y again on all cores
		
	   }
}




int main(void) {
	double x[Np]; double y[Np]; double omega[Nd];
        double ax[Np]; double ay[Np]; double angle[Nd];
        double E_pot[Nd]; // potential energy of the system
        
        double *dev_x, *dev_y, *dev_omega;
	double *dev_ax, *dev_ay, *dev_angle;
	double *dev_E_pot;

	hipMalloc( (void**)&dev_x,     Np*sizeof(double));
        hipMalloc( (void**)&dev_y,     Np*sizeof(double));
        hipMalloc( (void**)&dev_omega, Nd*sizeof(int));
	hipMalloc( (void**)&dev_ax,    Np*sizeof(double));
	hipMalloc( (void**)&dev_ay,    Np*sizeof(double));
	hipMalloc( (void**)&dev_angle, Nd*sizeof(double));
        hipMalloc( (void**)&dev_E_pot, Nd*sizeof(int));

	//TODO: init variables with Box-Muller and 2D gauss curve
        //      for two bodies with different centers and radia
	for (int i=0; i<Np; i++) {
		x[i] = (i*i)/1E6;
		y[i] = i/1000;
		ax[i] = 0;
		ay[i] = 0;
	}
	for (int i=0; i<Nd; i++) {
		omega[i] = 0;
		angle[i] = 0;
		E_pot[i] = 0;
	}

        hipMemcpy(dev_x, x, Np*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_y, y, Np*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_omega, omega, Nd*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_ax, x, Np*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_ay, x, Np*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_angle, x, Nd*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_E_pot, E_pot, Nd*sizeof(double), hipMemcpyHostToDevice);
	
	simulate_dipoles<<<Np,1>>>(dev_x, dev_y, dev_omega, dev_ax, dev_ay, dev_angle, dev_E_pot);
	
	hipMemcpy(x, dev_x, Np*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(y, dev_y, Np*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(omega, dev_omega, Nd*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(ax, dev_ax, Np*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(ay, dev_ay, Np*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(angle, dev_angle, Nd*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(E_pot, dev_E_pot, Nd*sizeof(double), hipMemcpyDeviceToHost);
	
	for (int i=0; i<Np; i++) {
		printf("i: %d ax: %g ay %g\n", i, ax[i], ay[i]);
	}
	
	hipFree(dev_x);
        hipFree(dev_y);
	hipFree(dev_omega);
	hipFree(dev_ax);
	hipFree(dev_ay);
	hipFree(dev_angle);
	hipFree(dev_E_pot);

}




