/* (c) 2016 Livio and Tiziano Mengotti */
/* n-dipoles simulation */
// to make it: nvcc n-dipoles.cu -o n-dipoles.out
// to execute the example ./n-dipoles.out

#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 640 // number of dipoles
#define t 0.0001 // timestamp
#define r 10e-15 // rutheford radius


__global__ void simulate_dipoles(double *x, double *y, double *omega, double *E_pot) {
	double ax[N*2]; double ay[N*2]; 
        	
	int tid = blockIdx.x; 
	//if (tid<N)
	//	c[tid] = a[tid] + b[tid];
}

int main(void) {
	double x[N*2]; double y[N*2]; 
	double omega[N];
        double E_pot[N]; // potential energy of the system

        double *dev_x, *dev_y, *dev_omega;
	double *dev_E_pot;

	hipMalloc( (void**)&dev_x,     N*2*sizeof(double));
        hipMalloc( (void**)&dev_y,     N*2*sizeof(double));
        hipMalloc( (void**)&dev_omega, N*sizeof(int));
        hipMalloc( (void**)&dev_E_pot, N*sizeof(int));

	for (int i=0; i<N*2; i++) {
		x[i] = -i,
		y[i] = i*i;
		omega[i] = 0;
		E_pot[i] = 0;
	}

        hipMemcpy(dev_x, x, N*2*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_y, y, N*2*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_omega, omega, N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_E_pot, E_pot, N*sizeof(double), hipMemcpyHostToDevice);
	
	simulate_dipoles<<<N,1>>>(dev_x, dev_y, dev_omega, dev_E_pot);

	hipMemcpy(omega, dev_omega, N*sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(E_pot, dev_E_pot, N*sizeof(double), hipMemcpyDeviceToHost);
	
	for (int i=0; i<N; i++) {
		printf("i: %d omega: %g E_pot %g\n", i, omega[i], E_pot[i]);
	}
	
	hipFree(dev_x);
        hipFree(dev_y);
	hipFree(dev_omega);
	hipFree(dev_E_pot);

}




